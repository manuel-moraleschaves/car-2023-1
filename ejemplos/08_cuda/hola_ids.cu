
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hola desde el bloque %d, hilo %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
}