#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

void helloCPU()
{
  std::cout<<"Hello from Cpu.\n";
}

__global__
void helloGPU()
{
  printf("Hello also from Gpu.\n");
}

int main()
{

  helloCPU();
  helloGPU<<<1,1>>>();
  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
