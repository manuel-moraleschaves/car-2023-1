#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  //float *x = new float[N];
  //float *y = new float[N];
  float *x;
  float *y;
  hipMallocManaged (&x, N);
  hipMallocManaged (&y, N);

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  add<<<number_of_blocks, threads_per_block>>>(N, x, y);
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}