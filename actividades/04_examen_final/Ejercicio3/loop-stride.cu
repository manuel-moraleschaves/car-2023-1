#include <cstdio>
#include "hip/hip_runtime.h"

void init(int *a, int N)
{
  for (int i = 0; i < N; ++i) {
    a[i] = i;
  }
}

/*
	Cree una función que duplique cada uno de los elementos del array a de forma paralela, cada hilo debe de trabajar sobre más de un elemento. No cambie el N ni el tamaño del grid de hilos
*/
__global__
void doubleElements(int *a, int N)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < N; i += blockDim.x*gridDim.x) {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  for (int i = 0; i < N; ++i) {
    if (a[i] != i*2) {
      return false;
    }
  }
  return true;
}

int main()
{
  /*
   * 'N' is greater than the size of the grid (see below).
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  /*
   * The size of this grid is 256*32 = 8192.
   */

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;

  printf("Starting...\n\n");
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
